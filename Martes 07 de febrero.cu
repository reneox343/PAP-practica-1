#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>

using namespace std;

#define GPUErrorAssertion(ans) {gpuAssert((ans), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n\r", hipGetErrorString(code), file, line);
        if (abort)exit(code);
    }
}

__global__ void matrix_mult(int* a, int* b, int* c, int size) {
    int row = threadIdx.x / size;
    int col = threadIdx.x - row * size;

    int suma = 0;
    if (row < size && col < size) {
        for (int i = 0; i < size; i++) {
            suma += a[row * size + i] * b[i * size + col];
        }
    }
    c[threadIdx.x] = suma;
}

int main() {

    const int width = 2;
    int* host_a, * host_b, * host_c;
    int* dev_a, * dev_b, * dev_c;
    host_a = (int*)malloc(width * width * sizeof(int));
    host_b = (int*)malloc(width * width * sizeof(int));
    host_c = (int*)malloc(width * width * sizeof(int));
    hipMalloc(&dev_a, width * width * sizeof(int));
    hipMalloc(&dev_b, width * width * sizeof(int));
    hipMalloc(&dev_c, width * width * sizeof(int));
    for (int i = 0; i < width * width; i++) {
        int r1 = (rand() % (256));
        int r2 = (rand() % (256));
        host_a[i] = r1;
        host_b[i] = r2;
        host_c[i] = 0;
    }
    hipMemcpy(dev_a, host_a, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, host_c, width * width * sizeof(int), hipMemcpyHostToDevice);

    matrix_mult << <1, 32 >> > (dev_a, dev_b, dev_c, width);
    hipMemcpy(host_c, dev_c, width * width * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipDeviceReset();

    cout << "A:\n";
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            cout << host_a[i * width + j] << " ";
        }
        cout << "\n";
    }
    cout << "B:\n";
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            cout << host_b[i * width + j] << " ";
        }
        cout << "\n";
    }

    cout << "C: \n";
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            cout << host_c[i * width + j] << " ";
        }
        cout << "\n";
    }
    free(host_a);
    free(host_b);
    free(host_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);


    return 0;
}